
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

int main(int argc, char **argv) {
    int N=atoi(argv[1]);;
    long long sum = 0; 
    double average;

    clock_t start_time = clock();

    for (int i = 1; i <= N; i++) {
        sum += i;
    }

    average = (double)sum / N;

    clock_t end_time = clock();
    double time_taken = ((double)(end_time - start_time)) / CLOCKS_PER_SEC * 1000.0; // Convert to milliseconds

    printf("%.2f", time_taken);

    return 0;
}
