
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

int main(int argc, char **argv) {
    int N=atoi(argv[1]);;
    long long sum = 0; 
    long long sum_of_squares = 0;
    double mean, stdev;
    
    clock_t start_time = clock();

    for (int i = 1; i <= N; i++) {
        sum += i;
        sum_of_squares += i * i;
    }

    mean = (double)sum / N;

    double variance = (double)sum_of_squares / N - mean * mean;
    stdev = sqrt(variance);

    clock_t end_time = clock();
    double time_taken = ((double)(end_time - start_time)) / CLOCKS_PER_SEC * 1000.0; // Convert to milliseconds

    printf("%.2f", time_taken);

    return 0;
}
